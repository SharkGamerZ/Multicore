/**
* Developed By Karan Bhagat
* February 2017
* compile with: nvcc image_stub.cu -o image_stub 
**/


#include <hip/hip_runtime.h>
#include <stdio.h>
//number of channels i.e. R G B
#define CHANNELS 3
#define BLUR_SIZE 4

unsigned char* loadPPM(const char* path, int* width, int* height);
void writePPM(const char* path, unsigned char* img, int width, int height);

__global__ 	void kernel_blur(unsigned char* d_rgb_image,unsigned char*d_blur_image,
                        int rows,int cols,int bsize) 
{
	int c = threadIdx.x+blockIdx.x*blockDim.x;
	int r = threadIdx.y+blockIdx.y*blockDim.y;
	
	if(c >= cols || r >= rows) return;

    unsigned int red  =0;
    unsigned int green=0;
    unsigned int blue =0;
    int num=0; 

    int curr_c;
	int curr_r;

	for (int i = -bsize; i <= bsize; i++)
		for (int j = -bsize; j <= bsize; j++) {
			curr_c = c + i;
			curr_r = r + j;
			if((curr_r<0)||(curr_r>rows-1)||(curr_c<0)||(curr_c>cols-1)) continue; 
			red   += d_rgb_image[(3*(curr_c+curr_r*cols))];
			green += d_rgb_image[(3*(curr_c+curr_r*cols))+1];
			blue  += d_rgb_image[(3*(curr_c+curr_r*cols))+2];
			num++;
			}
	red /= num;
	green /= num;
	blue /= num;


	d_blur_image[3*(c+r*cols)]	=red;
    d_blur_image[3*(c+r*cols)+1]=green;
    d_blur_image[3*(c+r*cols)+2]=blue;



}

int main(int argc, char **argv) 
{
	char* input_file;
	char* output_file;
    int bsize=4;
	//Check for the input file and output file names
	switch(argc) {
		case 4:
			input_file = argv[1];
			output_file = argv[2];
			bsize = atoi(argv[3]);
            break;
		case 3:
			input_file = argv[1];
			output_file = argv[2];
			bsize = 4;
            break;
		default:
			printf("Usage: <executable> input_file output_file bsize\n");
			exit(1);
	}
	
	unsigned char *h_rgb_image; //store image's rbg data
	unsigned char *d_rgb_image; //array for storing rgb data on device
	unsigned char *h_blur_image, *d_blur_image; //host and device's blur image data array pointers
	int rows; //number of rows of pixels
	int cols; //number of columns of pixels
	
	//load image into an array and retrieve number of pixels
	h_rgb_image = loadPPM(input_file, &cols, &rows); 

	int total_pixels=rows*cols;
	//allocate memory of host's blur image data array
	h_blur_image = (unsigned char *)malloc(sizeof(unsigned char*) * total_pixels * CHANNELS);

    

	/* ADD YOUR CODE HERE */
	//memcpy(h_blur_image,h_rgb_image,total_pixels*CHANNELS);
	hipMalloc(&d_rgb_image,total_pixels*CHANNELS*8);
	hipMalloc(&d_blur_image,total_pixels*CHANNELS*8);
	
	hipMemcpy(d_rgb_image,h_rgb_image,total_pixels*CHANNELS*8,hipMemcpyHostToDevice);
    dim3 BlockSize(16,16,1);
	dim3 GridSize((cols/16)+1,(rows/16)+1,1);
    	
	kernel_blur<<<GridSize,BlockSize>>>(d_rgb_image,d_blur_image,rows,cols,bsize);
    hipMemcpy(h_blur_image,d_blur_image,total_pixels*CHANNELS*8,hipMemcpyDeviceToHost);
	
	/* END OF YOUR CODE */

	//output the blurred image
    writePPM(output_file, h_blur_image, cols, rows);
	hipFree(d_rgb_image);
	hipFree(d_blur_image);
	return 0;
}

//function for loading an image into rgb format unsigned char array

unsigned char* loadPPM(const char* path, int* width, int* height) {
	FILE* file = fopen(path, "rb");

	if (!file) {
		fprintf(stderr, "Failed to open file\n");
		return NULL;
	}

	char header[3];
	fscanf(file, "%2s", header);
	if (header[0] != 'P' || header[1] != '6') {
		fprintf(stderr, "Invalid PPM file\n");
		return NULL;
	}

	fscanf(file, "%d %d", width, height);

	int maxColor;
	fscanf(file, "%d", &maxColor);

	fgetc(file);  // Skip single whitespace character

	unsigned char* img = (unsigned char*) malloc((*width) * (*height) * CHANNELS);
	if (!img) {
		fprintf(stderr, "Failed to allocate memory\n");
		return NULL;
	}

	fread(img, CHANNELS, *width * *height, file);

	fclose(file);

	return img;
}

//function for writing blurred image data array to the image file
void writePPM(const char* path, unsigned char* img, int width, int height) {
    FILE* file = fopen(path, "wb");

    if (!file) {
        fprintf(stderr, "Failed to open file\n");
        return;
    }

    fprintf(file, "P6\n%d %d\n255\n", width, height);

    fwrite(img, CHANNELS, width * height, file);

    fclose(file);
}